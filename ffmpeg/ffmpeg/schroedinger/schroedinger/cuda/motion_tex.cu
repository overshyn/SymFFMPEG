#include "hip/hip_runtime.h"
/** 
Overlapped block motion compensation for CUDA

Efficient implementation that divides up the image into regions based on the 
amount of blocks that overlap it, which is 1 (in the middle), 2 (horizontal 
or vertical overlap) or 4 (diagonal overlap). 

By processing these regions in different cuda blocks, any divergence between
threads is prevented.

*/
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>

// includes, project
//#include <cutil.h>
#include <hip/hip_runtime.h>

#include <algorithm>
#include <cassert>

#define SPECIAL

//#define USING_GLOBAL

#define THREADSX_LOG2 4
#define THREADSY_LOG2 4

#ifdef SPECIAL
#define WIDTHX_LOG2 (THREADSX_LOG2+1)
#define WIDTHY_LOG2 THREADSY_LOG2
#else
#define WIDTHX_LOG2 THREADSX_LOG2
#define WIDTHY_LOG2 THREADSY_LOG2
#endif

#define THREADSX (1<<THREADSX_LOG2)
#define THREADSY (1<<THREADSY_LOG2)

#define WIDTHX (1<<WIDTHX_LOG2)
#define WIDTHY (1<<WIDTHY_LOG2)

#include "common.h"
#include "cudamotion.h"

#include "motion_kernel_tex.h"

static inline int div_roundup(int x, int y)
{
    return (x+y-1)/y;
}

/// Private object
struct _CudaMotion
{
  hipStream_t stream;
  struct _MotionVector *vectors;
  struct hipArray *bdata;
};

__global__ void tex_copy(
    uint16_t* dest, int dstride, int dwidth, int dheight
)
{
  worktype2 val;
  int xofs = (blockIdx.x << WIDTHX_LOG2) + (threadIdx.x<<1);
  int yofs = (blockIdx.y << WIDTHY_LOG2) + threadIdx.y;
  dest = OFFSET_U16(dest, __mul24(dstride, yofs) + (xofs<<1));

  val.x = (worktype)(tex2D(ref1, xofs*2.0f+0.5f, yofs*2.0f+0.5f)*256.0f);
  val.y = (worktype)(tex2D(ref1, xofs*2.0f+3.5f, yofs*2.0f+0.5f)*256.0f);

  int16_2 sval;
  sval.x = val.x - 128;
  sval.y = val.y - 128;
  *((int16_2*)dest) = sval;
}

extern "C" {

CudaMotion *cuda_motion_init(hipStream_t stream)
{
    CudaMotion *rv;

    rv = new CudaMotion;
    rv->vectors = 0;
    rv->bdata = 0;
    rv->stream = stream;

    return rv;
}

void cuda_motion_free(CudaMotion *rv)
{
    hipFreeArray(rv->bdata);
    hipHostFree((void*)rv->vectors);

    delete rv;
}

struct _MotionVector *cuda_motion_reserve(CudaMotion *self, int width, int height)
{
    /// XXX check for dimension changes!
    hipChannelFormatDesc bdesc = hipCreateChannelDesc<short4>();
    if(!self->vectors)
        hipHostMalloc((void**)&self->vectors, width*height*sizeof(struct _MotionVector));
    if(!self->bdata)
        hipMallocArray(&self->bdata, &bdesc, width, height);
    return self->vectors;
}

void cuda_motion_begin(CudaMotion *self, CudaMotionData *d)
{
    /// Upload motion vectors
    hipChannelFormatDesc bdesc = hipCreateChannelDesc<short4>();

    hipMemcpy2DToArrayAsync(self->bdata, 0, 0, self->vectors, d->obmc.blocksx*8,
                     d->obmc.blocksx*8, d->obmc.blocksy, hipMemcpyHostToDevice, self->stream);

    bt1.addressMode[0] = hipAddressModeClamp;
    bt1.addressMode[1] = hipAddressModeClamp;
    bt1.filterMode = hipFilterModePoint;
    bt1.normalized = false;

    ref1.addressMode[0] = hipAddressModeClamp;
    ref1.addressMode[1] = hipAddressModeClamp;
    ref1.filterMode = hipFilterModeLinear;
    ref1.normalized = false;

    ref2.addressMode[0] = hipAddressModeClamp;
    ref2.addressMode[1] = hipAddressModeClamp;
    ref2.filterMode = hipFilterModeLinear;
    ref2.normalized = false;

    /// Bind motion vector texture
    hipBindTextureToArray(bt1, self->bdata, bdesc);
}

void cuda_motion_copy(CudaMotion *self, CudaMotionData *d, int16_t *output, int ostride, int width, int height, int component, int xshift, int yshift, struct hipArray *aref1, struct hipArray *aref2)
{
    /// Bind reference texture
    hipChannelFormatDesc channelDesc = hipCreateChannelDesc(8, 0, 0, 0, hipChannelFormatKindUnsigned);
    hipBindTextureToArray(ref1, aref1, channelDesc);
    if(aref2)
        hipBindTextureToArray(ref2, aref2, channelDesc);

    /// Execute the kernel
    dim3 block_size, grid_size;
    int shared_size;

    /// Determine execution parameters
    block_size.x = THREADSX;
    block_size.y = THREADSY;
    block_size.z = 1;

    int blocksX = div_roundup(width,  d->obmc.x_sep>>xshift);
    int blocksY = div_roundup(height, d->obmc.y_sep>>yshift);
    int xB = div_roundup(blocksX * (d->obmc.x_mid>>xshift), WIDTHX);
    int yB = div_roundup(blocksY * (d->obmc.y_mid>>yshift), WIDTHY);
    int xC = div_roundup((blocksX+1) * (d->obmc.x_ramp>>xshift), WIDTHX);
    int yC = div_roundup((blocksY+1) * (d->obmc.y_ramp>>yshift), WIDTHY);
/*
    printf("%i %i : %i %i %i %i %i %i : %i %i %i %i %i %i : %i %i\n", width, height, 
                                  d->obmc.x_sep>>xshift, d->obmc.y_sep>>yshift, d->obmc.x_mid>>xshift, d->obmc.y_mid>>yshift, d->obmc.x_ramp>>xshift, d->obmc.y_ramp>>yshift,
                                  blocksX, blocksY, xB, yB, xC, yC,
                                  xB + xC, yB + yB);
*/
    grid_size.x = xB + xC;
    grid_size.y = yB + yC;
    grid_size.z = 1;
    shared_size = 0;

/*
    printf("%ix%i comp %i grid %ix%i scale %fx%f ramp %i %i sep %i %i mid %i %i\n", 
        width, height,
        component, grid_size.x, grid_size.y, sxscale, syscale,
        d->obmc.x_ramp_log2 - xshift, d->obmc.y_ramp_log2 - yshift, 
        d->obmc.x_sep_log2 - xshift,  d->obmc.y_sep_log2 - yshift, 
        d->obmc.x_mid_log2 - xshift,  d->obmc.y_mid_log2 - yshift
    );
*/
#if 1
    if((d->obmc.x_ramp_log2 - xshift - 1) == 0)
    {
      /** Slower variant, use 2 byte writes */
      motion_copy_2ref_2b<<<grid_size, block_size, shared_size, self->stream>>>(
          output, ostride, width, height, xB, yB,
          component*8,
          d->obmc.x_ramp_log2 - xshift, d->obmc.y_ramp_log2 - yshift, 
          d->obmc.x_sep_log2 - xshift,  d->obmc.y_sep_log2 - yshift, 
          d->obmc.x_mid_log2 - xshift,  d->obmc.y_mid_log2 - yshift,
          d->obmc.weight1, d->obmc.weight2, d->obmc.weight_shift,
          xshift, yshift
          );
    }
    else
    {
      /** Fast variant, use 4 byte writes */
      motion_copy_2ref_4b<<<grid_size, block_size, shared_size, self->stream>>>(
          output, ostride, width, height, xB, yB,
          component*8,
          d->obmc.x_ramp_log2 - xshift, d->obmc.y_ramp_log2 - yshift, 
          d->obmc.x_sep_log2 - xshift,  d->obmc.y_sep_log2 - yshift, 
          d->obmc.x_mid_log2 - xshift,  d->obmc.y_mid_log2 - yshift,
          d->obmc.weight1, d->obmc.weight2, d->obmc.weight_shift,
          xshift, yshift
          );
    }
#else
    tex_copy<<<dim3(div_roundup(width,WIDTHX),div_roundup(height,WIDTHY),1), block_size, 0>>>(
        output, ostride, width, height);
#endif
    /// XXX unbind textures?
}


}
